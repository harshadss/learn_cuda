
#include <hip/hip_runtime.h>
#include<iostream>
#include<unistd.h>

__global__ void add(float* h_x, float* h_y, float* h_z, unsigned int N) {
    unsigned int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (index < N) {
        h_z[index] = h_x[index] + h_y[index];
    }
}

int main(void) {

    const unsigned int N = 1024;

    // create local data
    float* h_x = new float[N];
    float* h_y = new float[N];
    float* h_z = new float[N];

    for (auto i = 0; i < N; i++) {
        h_x[i] = 1.0;
        h_y[i] = 2.0;
    }

    // create pointers to hold device memory
    float* d_x; 
    float* d_y; 
    float* d_z;

    sleep(3);
    
    // allocate space on device
    hipMalloc((void**)&d_x, N*sizeof(float));
    hipMalloc((void**)&d_y, N*sizeof(float));
    hipMalloc((void**)&d_z, N*sizeof(float));
    // move local data to device
    hipMemcpy(d_x, h_x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, N*sizeof(float), hipMemcpyHostToDevice);
    // launch kernel
    // kernel config
    unsigned int numThreadsInBlock = 32;
    unsigned int numBlocks = (N + numThreadsInBlock - 1)/numThreadsInBlock;

    dim3 blockConfig {numThreadsInBlock};
    dim3 gridConfig {numBlocks};

    add<<<gridConfig, blockConfig>>>(d_x, d_y, d_z, N);

    hipDeviceSynchronize();
    // move back device to host
    hipMemcpy(h_z, d_z, N*sizeof(float), hipMemcpyDeviceToHost);
    sleep(3);
    // free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    sleep(3);
    // validate answer
    for (auto i = 0; i < 10; i++) {
        std::cout << "i: " << i << "\th_z[i]: " << h_z[i] << std::endl;
    }
    // free host memory
    delete[] h_x;
    delete[] h_y;
    delete[] h_z;
    return 0;
}